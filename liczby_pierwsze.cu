#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/reduce.h>
#include <iostream>
#include <cmath>
#include <cstdio>
#ifndef __HIPCC__ 
#define __HIPCC__
#endif
#include <hip/device_functions.h>

//kernel ustawiajacy co x-ta pozycje w tablicy na 0 (FALSE) od x elementu
__global__ void falseFlagsMaker(unsigned int *container, unsigned int containerSize, unsigned int x) {
	unsigned int threadPos = threadIdx.x + blockIdx.x * blockDim.x;
	if (threadPos < containerSize) {
		if (threadPos == 0 || threadPos == 1) {
			container[threadPos] = 0;
		}
		if (threadPos%x == 0 && threadPos != x) {
			container[threadPos] = 0;
		}
	}
}

//kernel przygotowywujacy tablice pod operacje reduce do znalezienia kolejnej liczby do testowania
__global__ void makeArray(unsigned int *container, unsigned int *temp, unsigned int size) {
	unsigned int threadPos = threadIdx.x + blockIdx.x * blockDim.x;
	if (threadPos < size) {
		if (container[threadPos] == 0) temp[threadPos] = size;
		else if (container[threadPos] == 1) temp[threadPos] = threadPos;
	}
}

//struktura, model predykatu - potrzebny do operacji copy_if
struct is_one
{
	__host__ __device__
		bool operator()(const int x)
	{
		return x == 1;
	}
};

int main() {
	std::cout << " ----- Projekt 3: Szukanie liczb pierwszych ----- " << std::endl << std::endl;

	// podanie maksymalnej sprawdzanej liczby
	unsigned int m;
	std::cout << "Podaj liczbe, do ktorej mam szukac liczb pierwszych: ";
	std::cin >> m;

	if (m < 1) {
		std::cout << "BRAK LICZB PIERWSZYCH" << std::endl;
		return 0;
	}
	else if(m==2){
		std::cout << 2 << std::endl;
		return 0;
	}
	else {
		// wielkosc wektora flags
		unsigned const int ARRAY_SIZE = m + 1;

		// zaalokownie wektora hosta
		thrust::host_vector<unsigned int> hostFlags(ARRAY_SIZE);
		//wypelnienie wektora hosta
		for (int i = 0; i < ARRAY_SIZE; i++) {
			hostFlags[i] = 1;
		}
		//kopiowanie host -> device
		thrust::device_vector<unsigned int> deviceFlags = hostFlags;

		//zaalokowanie i inicializacja zerami wektorow pomocniczych
		thrust::device_vector<unsigned int> deviceTemp(ARRAY_SIZE);
		thrust::host_vector<unsigned int> hostTemp = deviceTemp;

		//2 jest pierwsza liczba pierwsza, 2 jest pod indeksem 2
		int index = 2;

		//przesiewamy do pierwiastka z m - Sito Erastotenesa
		int stop = ceil(sqrt(m));
		while (index <= stop) {
			falseFlagsMaker << <1, ARRAY_SIZE >> > (deviceFlags.data().get(), ARRAY_SIZE, index);
			makeArray << <1, ARRAY_SIZE >> > (deviceFlags.data().get(), deviceTemp.data().get(), ARRAY_SIZE);
			index = thrust::reduce(deviceTemp.begin() + (index + 1), deviceTemp.end(), -1, thrust::minimum<unsigned int>());
			if (index > stop) {
				hostTemp = deviceTemp;
				hostFlags = deviceFlags;
			}
		}

		//jak duza ma byc tablica wynikowa
		int amount = 0;
		for (int i = 0; i < ARRAY_SIZE; i++) {
			if (hostFlags[i] == 1) {
				amount++;
			}
		}

		//wektory wynikowe
		thrust::device_vector<unsigned int> deviceResult(amount);
		thrust::host_vector<unsigned int> hostResult;

		//copy_if
		thrust::copy_if(deviceTemp.begin(), deviceTemp.end(), deviceFlags.begin(), deviceResult.begin(), is_one());
		hostResult = deviceResult;

		//wyswietlanie wyniku
		for (int i = 0; i < hostResult.size(); i++) {
			std::cout << hostResult[i] << ", ";
		}
		std::cout << std::endl;

		return 0;
	}

	return 0;
}